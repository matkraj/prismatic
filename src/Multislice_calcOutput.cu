#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// Prismatic is distributed under the GNU General Public License (GPL)
// If you use Prismatic, we kindly ask that you cite the following papers:

// 1. Ophus, C.: A fast image simulation algorithm for scanning
//    transmission electron microscopy. Advanced Structural and
//    Chemical Imaging 3(1), 13 (2017)

// 2. Pryor, Jr., A., Ophus, C., and Miao, J.: A Streaming Multi-GPU
//    Implementation of Image Simulation Algorithms for Scanning
//	  Transmission Electron Microscopy. arXiv:1706.08563 (2017)

// / Calculate result of Multislice simulation using GPU and (potentially) CPU. Multiple GPU threads are launched, each with
// their own memory buffers. Page-locked host memory is allocated so that memory transfers to the GPU can occur asynchronously,
// and memory allocation for the GPU occurs only once, as each call to hipMalloc will potentially interrupt concurrent execution.
// Each GPU/CPU worker thread repeatedly calls getWorkID to be assigned probe positions to compute. This queue mechanism
// ensures that both the CPU and GPU are kept busy.

// For variable naming, the suffixes are "_d" for "device" (1 copy per GPU), "_ds" for "device stream (1 copy per stream), "_ph" for "pinned host"

#include "Multislice_calcOutput.cuh"
#include "Multislice_calcOutput.h"
#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"
#include "utility.cuh"
#include "params.cuh"

namespace Prismatic{
	extern std::mutex fftw_plan_lock;

	inline void createPlansAndStreamsM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
									   CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		// create CUDA streams
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;
		cuda_pars.streams   = new hipStream_t[total_num_streams];
		cuda_pars.cufftPlans = new hipfftHandle[total_num_streams];

		// batch parameters for cuFFT
		const int rank = 2;
		int n[] = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
		const int howmany = pars.meta.batchSizeGPU;
		int idist = n[0]*n[1];
		int odist = n[0]*n[1];
		int istride = 1;
		int ostride = 1;
		int *inembed = n;
		int *onembed = n;

		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.numGPUs);
			cudaErrchk(hipStreamCreate(&cuda_pars.streams[j]));
			cufftErrchk(hipfftPlanMany(&cuda_pars.cufftPlans[j], rank, n, inembed, istride, idist, onembed, ostride, odist, PRISMATIC_CUFFT_PLAN_TYPE, howmany));
			cufftErrchk(hipfftSetStream(cuda_pars.cufftPlans[j], cuda_pars.streams[j]));
		}
	}

	inline void allocatePinnedHostMemory_M(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                       CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;
		cuda_pars.output_ph = new PRISMATIC_FLOAT_PRECISION*[total_num_streams];
		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.PsiProbeInit_ph, pars.psiProbeInit.size()*sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.trans_ph,        pars.transmission.size()*sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.prop_ph,         pars.prop.size()*sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.qxa_ph,          pars.qxa.size()*sizeof(PRISMATIC_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.qya_ph,          pars.qya.size()*sizeof(PRISMATIC_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&cuda_pars.alphaInd_ph,     pars.alphaInd.size()*sizeof(PRISMATIC_FLOAT_PRECISION)));
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &cuda_pars.output_ph[s], pars.output.get_dimi() * sizeof(PRISMATIC_FLOAT_PRECISION)));
		}
	}

	inline void copyToPinnedMemory_M(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                          CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		// copy host memory to pinned
		memcpy(cuda_pars.PsiProbeInit_ph, &pars.psiProbeInit[0], pars.psiProbeInit.size() * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>));
		memcpy(cuda_pars.trans_ph,        &pars.transmission[0], pars.transmission.size() * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>));
		memcpy(cuda_pars.prop_ph,         &pars.prop[0],         pars.prop.size()         * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>));
		memcpy(cuda_pars.qxa_ph,          &pars.qxa[0],          pars.qxa.size()          * sizeof(PRISMATIC_FLOAT_PRECISION));
		memcpy(cuda_pars.qya_ph,          &pars.qya[0],          pars.qya.size()          * sizeof(PRISMATIC_FLOAT_PRECISION));
		memcpy(cuda_pars.alphaInd_ph,     &pars.alphaInd[0],     pars.alphaInd.size()     * sizeof(PRISMATIC_FLOAT_PRECISION));
	}

	inline void allocateDeviceMemory_singlexferM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                             CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;

		// pointers to read-only GPU memory (one copy per GPU)
		cuda_pars.PsiProbeInit_d  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[pars.meta.numGPUs];
		cuda_pars.trans_d		  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[pars.meta.numGPUs];
		cuda_pars.prop_d 		  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[pars.meta.numGPUs];
		cuda_pars.qxa_d 		  = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];
		cuda_pars.qya_d 		  = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];
		cuda_pars.alphaInd_d      = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];

		// pointers to read/write GPU memory (one per stream)
		cuda_pars.psi_ds 			  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.psiIntensity_ds    = new PRISMATIC_FLOAT_PRECISION*[total_num_streams];
		cuda_pars.integratedOutput_ds = new PRISMATIC_FLOAT_PRECISION*[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.numGPUs; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &cuda_pars.PsiProbeInit_d[g],     pars.psiProbeInit.size()   * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.trans_d[g],            pars.transmission.size()   * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.prop_d[g],             pars.prop.size()           * sizeof(PRISMATIC_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qxa_d[g],              pars.qxa.size()            * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qya_d[g],              pars.qya.size()            * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.alphaInd_d[g],         pars.alphaInd.size()       * sizeof(PRISMATIC_FLOAT_PRECISION)));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.numGPUs));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_ds[s],              pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psiIntensity_ds[s],     pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.integratedOutput_ds[s], pars.detectorAngles.size()                        * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(cuda_pars.psi_ds[s], 0,                      pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(std::complex<PRISMATIC_FLOAT_PRECISION>)));
			cudaErrchk(hipMemset(cuda_pars.psiIntensity_ds[s], 0,             pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(cuda_pars.integratedOutput_ds[s], 0,         pars.detectorAngles.size()                        * sizeof(PRISMATIC_FLOAT_PRECISION)));
		}
	}

	inline void allocateDeviceMemory_streamingM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                            CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;

		// pointers to read-only GPU memory (one copy per GPU)
		cuda_pars.PsiProbeInit_d  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[pars.meta.numGPUs];
		cuda_pars.prop_d 	   	  = new PRISMATIC_CUDA_COMPLEX_FLOAT*[pars.meta.numGPUs];
		cuda_pars.qxa_d 		  = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];
		cuda_pars.qya_d 		  = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];
		cuda_pars.alphaInd_d 	  = new PRISMATIC_FLOAT_PRECISION*[pars.meta.numGPUs];

		// pointers to read/write GPU memory (one per stream)
		cuda_pars.trans_d 		      = new PRISMATIC_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.psi_ds  		      = new PRISMATIC_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.psiIntensity_ds    = new PRISMATIC_FLOAT_PRECISION*[total_num_streams];
		cuda_pars.integratedOutput_ds = new PRISMATIC_FLOAT_PRECISION*[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.numGPUs; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &cuda_pars.PsiProbeInit_d[g],     pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.prop_d[g],             pars.prop.size()         * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qxa_d[g],              pars.qxa.size()          * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qya_d[g],              pars.qya.size()          * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.alphaInd_d[g],         pars.alphaInd.size()     * sizeof(pars.alphaInd[0])));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.numGPUs));
			cudaErrchk(hipMalloc((void **) &cuda_pars.trans_d[s],             pars.transmission.get_dimj() * pars.transmission.get_dimi() * sizeof(pars.transmission[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_ds[s],              pars.meta.batchSizeGPU*pars.psiProbeInit.size()           * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psiIntensity_ds[s],    pars.meta.batchSizeGPU*pars.psiProbeInit.size()           * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.integratedOutput_ds[s], pars.detectorAngles.size()                                  * sizeof(PRISMATIC_FLOAT_PRECISION)));

			cudaErrchk(hipMemset(cuda_pars.psi_ds[s],              0, pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMemset(cuda_pars.psiIntensity_ds[s],    0, pars.meta.batchSizeGPU*pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(cuda_pars.integratedOutput_ds[s], 0, pars.detectorAngles.size()                        * sizeof(PRISMATIC_FLOAT_PRECISION)));
		}
	}

	inline void copyToGPUMemory_singlexferM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                        CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){

		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;

		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.numGPUs; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(cuda_pars.PsiProbeInit_d[g], &cuda_pars.PsiProbeInit_ph[0], pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.trans_d[g], &cuda_pars.trans_ph[0], pars.transmission.size() * sizeof(pars.transmission[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.prop_d[g], &cuda_pars.prop_ph[0], pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qxa_d[g], &cuda_pars.qxa_ph[0], pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qya_d[g], &cuda_pars.qya_ph[0], pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.alphaInd_d[g], &cuda_pars.alphaInd_ph[0], pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.numGPUs; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}
	}

	inline void copyToGPUMemory_streamingM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                       CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;

		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.numGPUs; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(cuda_pars.PsiProbeInit_d[g], &cuda_pars.PsiProbeInit_ph[0], pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.prop_d[g], &cuda_pars.prop_ph[0], pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qxa_d[g], &cuda_pars.qxa_ph[0], pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qya_d[g], &cuda_pars.qya_ph[0], pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.numGPUs) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.alphaInd_d[g], &cuda_pars.alphaInd_ph[0], pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.numGPUs; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}
	}

	inline void launchWorkers_singlexferM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                      CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;

		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations
		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
		const size_t PRISMATIC_PRINT_FREQUENCY_PROBES = max((size_t)1,pars.xp.size() * pars.yp.size() / 10); // for printing status
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size());

		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.numGPUs; // determine which GPU handles this job
			hipStream_t& current_stream = cuda_pars.streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " on GPU #" << GPU_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = cuda_pars.PsiProbeInit_d[GPU_num];
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_trans_d        = cuda_pars.trans_d[GPU_num];
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_prop_d         = cuda_pars.prop_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_qxa_d             = cuda_pars.qxa_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_qya_d             = cuda_pars.qya_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_alphaInd_d        = cuda_pars.alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_psi_ds           = cuda_pars.psi_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_psiIntensity_ds    = cuda_pars.psiIntensity_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_integratedOutput_ds = cuda_pars.integratedOutput_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_output_ph           = cuda_pars.output_ph[stream_count];
			hipfftHandle & current_cufft_plan                   = cuda_pars.cufftPlans[stream_count];

			// launch a new thread
			workers_GPU.push_back(thread([&pars, current_trans_d, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                             current_psi_ds, current_psiIntensity_ds, current_integratedOutput_ds,
					                             GPU_num, current_qya_d, current_qxa_d, current_output_ph, &current_cufft_plan,
					                             current_prop_d, &current_stream, &psi_size, stream_count, &PRISMATIC_PRINT_FREQUENCY_PROBES, &cuda_pars]() {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU

#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(Prismatic::memLock);
					size_t free_mem, total_mem;
					free_mem=total_mem=0;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.maxGPUMem = std::max(total_mem - free_mem, pars.maxGPUMem);
				}
#endif // NDEBUG
				size_t Nstart, Nstop;
				Nstart=Nstop=0;
				while (dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeGPU)){ // synchronously get work assignment
					while (Nstart < Nstop){
						if (Nstart % PRISMATIC_PRINT_FREQUENCY_PROBES < pars.meta.batchSizeGPU | Nstart == 100){
							cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << '\n';
						}
//							getMultisliceProbe_GPU_singlexfer(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, current_output_ph,
//							                                  current_psiIntensity_ds,
//							                                  current_integratedOutput_ds, current_qya_d, current_qxa_d,
//							                                  current_prop_d, ay, ax, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
//							                                  current_alphaInd_d, current_cufft_plan, current_stream);
						getMultisliceProbe_GPU_singlexfer_batch(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, current_output_ph,
						                                        current_psiIntensity_ds,
						                                        current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                                        current_prop_d, Nstart, Nstop, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
						                                        current_alphaInd_d, current_cufft_plan, current_stream);
#ifdef PRISMATIC_BUILDING_GUI
						pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
						Nstart=Nstop;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));
			++stream_count;
		}
		// now launch CPU work
		if (pars.meta.alsoDoCPUWork){
			PRISMATIC_FFTW_INIT_THREADS();
			PRISMATIC_FFTW_PLAN_WITH_NTHREADS(pars.meta.numThreads);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.numThreads); // prevents multiple reallocations
			// If the batch size is too big, the work won't be spread over the threads, which will usually hurt more than the benefit
			// of batch FFT
			pars.meta.batchSizeCPU = min(pars.meta.batchSizeTargetCPU, max((size_t)1, pars.xp.size() * pars.yp.size() / pars.meta.numThreads));
			cout << "multislice pars.meta.batchSizeGPU = " << pars.meta.batchSizeGPU << endl;
			for (auto t = 0; t < pars.meta.numThreads; ++t) {
				cout << "Launching CPU worker #" << t << endl;
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t, &PRISMATIC_PRINT_FREQUENCY_PROBES]() {
					size_t Nstart, Nstop, early_CPU_stop;
					Nstart=Nstop=0;
					// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
					// wait longer for everything to complete
					if (pars.meta.numGPUs > 0){
						// if there are no GPUs, make sure to do all work on CPU
						early_CPU_stop = (size_t)std::max((PRISMATIC_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.earlyCPUStopCount);
					} else {
						early_CPU_stop = pars.xp.size() * pars.yp.size();
					}
					if (dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeCPU, early_CPU_stop)) { // synchronously get work assignment
						Array1D<std::complex<PRISMATIC_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISMATIC_FLOAT_PRECISION> >({{pars.psiProbeInit.size() * pars.meta.batchSizeCPU}});

						// setup batch FFTW parameters
						const int rank    = 2;
						int n[]           = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
						const int howmany = pars.meta.batchSizeCPU;
						int idist         = n[0]*n[1];
						int odist         = n[0]*n[1];
						int istride       = 1;
						int ostride       = 1;
						int *inembed      = n;
						int *onembed      = n;
						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISMATIC_FFTW_PLAN plan_forward = PRISMATIC_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_FORWARD, FFTW_MEASURE);
						PRISMATIC_FFTW_PLAN plan_inverse = PRISMATIC_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_BACKWARD, FFTW_MEASURE);
						gatekeeper.unlock();

						// main work loop
						do {
							while (Nstart < Nstop) {
								if (Nstart % PRISMATIC_PRINT_FREQUENCY_PROBES  < pars.meta.batchSizeCPU | Nstart == 100){
									cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
								}
//							getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
								getMultisliceProbe_CPU_batch(pars, Nstart, Nstop, plan_forward, plan_inverse, psi_stack);
#ifdef PRISMATIC_BUILDING_GUI
								pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
								Nstart=Nstop;
							}
							if (Nstop >= early_CPU_stop) break;
						} while(dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeCPU, early_CPU_stop));
						gatekeeper.lock();
						PRISMATIC_FFTW_DESTROY_PLAN(plan_forward);
						PRISMATIC_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
				}));
			}
			cout << "Waiting on CPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISMATIC_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();

		for (auto g = 0; g < pars.meta.numGPUs; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}
	}

	inline void launchWorkers_streamingM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                                     CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;
		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations
		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
		const size_t PRISMATIC_PRINT_FREQUENCY_PROBES = max((size_t)1,pars.xp.size() * pars.yp.size() / 10); // for printing status
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size());
		// If the batch size is too big, the work won't be spread over the threads, which will usually hurt more than the benefit
		// of batch FFT

		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.numGPUs; // determine which GPU handles this job
			hipStream_t& current_stream = cuda_pars.streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << endl;

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d   = cuda_pars.PsiProbeInit_d[GPU_num];
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_prop_d           = cuda_pars.prop_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_qxa_d               = cuda_pars.qxa_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_qya_d               = cuda_pars.qya_d[GPU_num];
			PRISMATIC_FLOAT_PRECISION *current_alphaInd_d          = cuda_pars.alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_trans_ds         = cuda_pars.trans_d[stream_count];
			PRISMATIC_CUDA_COMPLEX_FLOAT *current_psi_ds           = cuda_pars.psi_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_psiIntensity_ds    = cuda_pars.psiIntensity_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_integratedOutput_ds = cuda_pars.integratedOutput_ds[stream_count];
			PRISMATIC_FLOAT_PRECISION *current_output_ph           = cuda_pars.output_ph[stream_count];
			hipfftHandle & current_cufft_plan                   = cuda_pars.cufftPlans[stream_count];
			// launch a new thread
			// push_back is better whenever constructing a new object
			workers_GPU.push_back(thread([&pars, current_trans_ds, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                             current_psi_ds, current_psiIntensity_ds, current_integratedOutput_ds,
					                             GPU_num, current_qya_d, current_qxa_d, current_output_ph, current_cufft_plan,
					                             current_prop_d, &current_stream, &psi_size, stream_count, &PRISMATIC_PRINT_FREQUENCY_PROBES, &cuda_pars]()  {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU


#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(Prismatic::memLock);
					size_t free_mem, total_mem;
					free_mem=total_mem=0;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.maxGPUMem = std::max(total_mem - free_mem, pars.maxGPUMem);
				}
#endif // NDEBUG

				size_t Nstart,Nstop;
				Nstart=Nstop=0;
				while (dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeGPU)){ // synchronously get work assignment
					while (Nstart < Nstop){
						if (Nstart % PRISMATIC_PRINT_FREQUENCY_PROBES < pars.meta.batchSizeGPU | Nstart == 100){
							cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
						}
						getMultisliceProbe_GPU_streaming_batch(pars, current_trans_ds, cuda_pars.trans_ph, current_PsiProbeInit_d, current_psi_ds,
						                                       current_output_ph, current_psiIntensity_ds,
						                                       current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                                       current_prop_d, Nstart, Nstop, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
						                                       current_alphaInd_d, current_cufft_plan, current_stream);
#ifdef PRISMATIC_BUILDING_GUI
						pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
						Nstart = Nstop;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));
			++stream_count;
		}

		// now launch CPU work
		if (pars.meta.alsoDoCPUWork){
			PRISMATIC_FFTW_INIT_THREADS();
			PRISMATIC_FFTW_PLAN_WITH_NTHREADS(pars.meta.numThreads);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.numThreads); // prevents multiple reallocations
			for (auto t = 0; t < pars.meta.numThreads; ++t) {
				cout << "Launching CPU worker #" << t << endl;
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t, &PRISMATIC_PRINT_FREQUENCY_PROBES]() {
					size_t Nstart, Nstop, early_CPU_stop;
					Nstart=Nstop=0;
					// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
					// wait longer for everything to complete
					if (pars.meta.numGPUs > 0){
						// if there are no GPUs, make sure to do all work on CPU
						early_CPU_stop = (size_t)std::max((PRISMATIC_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.earlyCPUStopCount);
					} else {
						early_CPU_stop = pars.xp.size() * pars.yp.size();
					}
					if (dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeCPU, early_CPU_stop)) { // synchronously get work assignment
						Array1D<std::complex<PRISMATIC_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISMATIC_FLOAT_PRECISION> >({{pars.psiProbeInit.size() * pars.meta.batchSizeCPU}});

						// setup batch FFTW parameters
						const int rank = 2;
						int n[] = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
						const int howmany = pars.meta.batchSizeCPU;
						int idist = n[0]*n[1];
						int odist = n[0]*n[1];
						int istride = 1;
						int ostride = 1;
						int *inembed = n;
						int *onembed = n;
						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISMATIC_FFTW_PLAN plan_forward = PRISMATIC_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_FORWARD, FFTW_MEASURE);
						PRISMATIC_FFTW_PLAN plan_inverse = PRISMATIC_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISMATIC_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_BACKWARD, FFTW_MEASURE);
						gatekeeper.unlock();

						// main work loop
						do {
							while (Nstart < Nstop) {
								if (Nstart % PRISMATIC_PRINT_FREQUENCY_PROBES  < pars.meta.batchSizeCPU | Nstart == 100){
									cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
								}
//							getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
								getMultisliceProbe_CPU_batch(pars, Nstart, Nstop, plan_forward, plan_inverse, psi_stack);
#ifdef PRISMATIC_BUILDING_GUI
								pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
								Nstart=Nstop;
							}
							if (Nstop >= early_CPU_stop) break;
						} while(dispatcher.getWork(Nstart, Nstop, pars.meta.batchSizeCPU, early_CPU_stop));
						gatekeeper.lock();
						PRISMATIC_FFTW_DESTROY_PLAN(plan_forward);
						PRISMATIC_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
					cout << "CPU worker #" << t << " finished\n";
				}));
			}
			cout << "Waiting on GPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISMATIC_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();

		for (auto g = 0; g < pars.meta.numGPUs; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}
	}

	inline void cleanupMemoryM(Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
	                           CudaParameters<PRISMATIC_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.numGPUs * pars.meta.numStreamsPerGPU;
		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.numGPUs; ++j){
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipFree(cuda_pars.PsiProbeInit_d[j]));
			cudaErrchk(hipFree(cuda_pars.trans_d[j]));
			cudaErrchk(hipFree(cuda_pars.qxa_d[j]));
			cudaErrchk(hipFree(cuda_pars.qya_d[j]));
			cudaErrchk(hipFree(cuda_pars.prop_d[j]));
			cudaErrchk(hipFree(cuda_pars.alphaInd_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.numGPUs));
			cudaErrchk(hipFree(cuda_pars.psi_ds[s]));
			cudaErrchk(hipFree(cuda_pars.psiIntensity_ds[s]));
			cudaErrchk(hipFree(cuda_pars.integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cuda_pars.cufftPlans[s]));
		}

		// free pinned memory
		cudaErrchk(hipHostFree(cuda_pars.PsiProbeInit_ph));
		cudaErrchk(hipHostFree(cuda_pars.trans_ph));
		cudaErrchk(hipHostFree(cuda_pars.prop_ph));
		cudaErrchk(hipHostFree(cuda_pars.qxa_ph));
		cudaErrchk(hipHostFree(cuda_pars.qya_ph));
		cudaErrchk(hipHostFree(cuda_pars.alphaInd_ph));
		for (auto s =0; s < total_num_streams; ++s){
			cudaErrchk(hipHostFree(cuda_pars.output_ph[s]));
		}

		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.numGPUs);
			cudaErrchk(hipStreamDestroy(cuda_pars.streams[j]));
		}
		for (auto j = 0; j < pars.meta.numGPUs; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
		delete[] cuda_pars.streams;
		delete[] cuda_pars.cufftPlans;
		delete[] cuda_pars.PsiProbeInit_d;
		delete[] cuda_pars.trans_d;
		delete[] cuda_pars.prop_d;
		delete[] cuda_pars.qxa_d;
		delete[] cuda_pars.qya_d;
		delete[] cuda_pars.alphaInd_d;
		delete[] cuda_pars.psi_ds;
		delete[] cuda_pars.psiIntensity_ds;
		delete[] cuda_pars.integratedOutput_ds;
		delete[] cuda_pars.output_ph;
	}

	// computes the result of probe position ay,ax using the GPU. The effect of this function is the same as getMultisliceProbe_CPU
	__host__ void getMultisliceProbe_GPU_singlexfer(Parameters<PRISMATIC_FLOAT_PRECISION>& pars,
	                                                PRISMATIC_CUDA_COMPLEX_FLOAT* trans_d,
	                                                PRISMATIC_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                PRISMATIC_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                PRISMATIC_FLOAT_PRECISION* output_ph,
	                                                PRISMATIC_FLOAT_PRECISION* psiIntensity_ds,
	                                                PRISMATIC_FLOAT_PRECISION* integratedOutput_ds,
									                const PRISMATIC_FLOAT_PRECISION* qya_d,
									                const PRISMATIC_FLOAT_PRECISION* qxa_d,
									                const PRISMATIC_CUDA_COMPLEX_FLOAT* prop_d,
									                const size_t ay,
									                const size_t ax,
									                const size_t dimj,
									                const size_t dimi,
									                const PRISMATIC_FLOAT_PRECISION* alphaInd_d,
									                const hipfftHandle& plan,
									                hipStream_t& stream){

		// initialize psi
		PRISMATIC_FLOAT_PRECISION yp = pars.yp[ay];
		PRISMATIC_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t psi_size = dimj*dimi;
		initializePsi<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, psi_size, yp, xp);
		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, &trans_d[planeNum*psi_size], psi_size);
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, psi_size);
			divide_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISMATIC_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
		}
		abs_squared<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
		formatOutput_GPU_integrate(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		real<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
		formatOutput_GPU_real(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		imag<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
		formatOutput_GPU_imag(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
}

	__host__ void getMultisliceProbe_GPU_singlexfer_batch(Parameters<PRISMATIC_FLOAT_PRECISION>& pars,
	                                                      PRISMATIC_CUDA_COMPLEX_FLOAT* trans_d,
	                                                      PRISMATIC_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                      PRISMATIC_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                      PRISMATIC_FLOAT_PRECISION* output_ph,
	                                                      PRISMATIC_FLOAT_PRECISION* psiIntensity_ds,
	                                                      PRISMATIC_FLOAT_PRECISION* integratedOutput_ds,
	                                                      const PRISMATIC_FLOAT_PRECISION* qya_d,
	                                                      const PRISMATIC_FLOAT_PRECISION* qxa_d,
	                                                      const PRISMATIC_CUDA_COMPLEX_FLOAT* prop_d,
	                                                      const size_t Nstart,
	                                                      const size_t Nstop,
	                                                      const size_t dimj,
	                                                      const size_t dimi,
	                                                      const PRISMATIC_FLOAT_PRECISION* alphaInd_d,
	                                                      const hipfftHandle& plan,
	                                                      hipStream_t& stream){
		const size_t psi_size = dimj*dimi;
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();

			// initialize psi
			PRISMATIC_FLOAT_PRECISION yp = pars.yp[ay];
			PRISMATIC_FLOAT_PRECISION xp = pars.xp[ax];

			initializePsi << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
                (psi_ds + (batch_idx * psi_size), PsiProbeInit_d, qya_d, qxa_d, psi_size, yp, xp);
		}
		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * psi_size), &trans_d[planeNum * psi_size], psi_size);
			}
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * psi_size), prop_d, psi_size);
				divide_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * psi_size), PRISMATIC_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
			}
		}
		
		abs_squared << < ( psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			formatOutput_GPU_integrate(pars, psiIntensity_ds + (batch_idx * psi_size),
			                           alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		}
		if (pars.meta.save4DComplexOutput) {
            real << < ( psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
            for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
                const size_t ay = (Nstart + batch_idx) / pars.xp.size();
                const size_t ax = (Nstart + batch_idx) % pars.xp.size();
                formatOutput_GPU_real(pars, psiIntensity_ds + (batch_idx * psi_size),
                                        alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
            }
            
            imag << < ( psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
            for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
                const size_t ay = (Nstart + batch_idx) / pars.xp.size();
                const size_t ax = (Nstart + batch_idx) % pars.xp.size();
                formatOutput_GPU_imag(pars, psiIntensity_ds + (batch_idx * psi_size),
                                        alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
            }
        }
	}

	__host__ void getMultisliceProbe_GPU_streaming(Parameters<PRISMATIC_FLOAT_PRECISION>& pars,
	                                               PRISMATIC_CUDA_COMPLEX_FLOAT* trans_d,
	                                               const complex<PRISMATIC_FLOAT_PRECISION>* trans_ph,
	                                               PRISMATIC_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                               PRISMATIC_CUDA_COMPLEX_FLOAT* psi_ds,
	                                               PRISMATIC_FLOAT_PRECISION* output_ph,
	                                               PRISMATIC_FLOAT_PRECISION* psiIntensity_ds,
	                                               PRISMATIC_FLOAT_PRECISION* integratedOutput_ds,
	                                               const PRISMATIC_FLOAT_PRECISION* qya_d,
	                                               const PRISMATIC_FLOAT_PRECISION* qxa_d,
	                                               const PRISMATIC_CUDA_COMPLEX_FLOAT* prop_d,
	                                               const size_t& ay,
	                                               const size_t& ax,
	                                               const size_t dimj,
	                                               const size_t dimi,
	                                               const PRISMATIC_FLOAT_PRECISION* alphaInd_d,
	                                               const hipfftHandle& plan,
	                                               hipStream_t& stream){
		// initialize psi
		PRISMATIC_FLOAT_PRECISION yp = pars.yp[ay];
		PRISMATIC_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t psi_size = dimj*dimi;
		initializePsi<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);


		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*psi_size], psi_size * sizeof(PRISMATIC_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, trans_d, psi_size);
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, psi_size);
			divide_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISMATIC_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
		}
		abs_squared<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
		formatOutput_GPU_integrate(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
        if (pars.meta.save4DComplexOutput) {
            real<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
            formatOutput_GPU_real(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
            imag<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psiIntensity_ds, psi_ds, psi_size);
            formatOutput_GPU_imag(pars, psiIntensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
        }

	}


	__host__ void getMultisliceProbe_GPU_streaming_batch(Parameters<PRISMATIC_FLOAT_PRECISION>& pars,
	                                                     PRISMATIC_CUDA_COMPLEX_FLOAT* trans_d,
	                                                     const complex<PRISMATIC_FLOAT_PRECISION>* trans_ph,
	                                                     PRISMATIC_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                     PRISMATIC_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                     PRISMATIC_FLOAT_PRECISION* output_ph,
	                                                     PRISMATIC_FLOAT_PRECISION* psiIntensity_ds,
	                                                     PRISMATIC_FLOAT_PRECISION* integratedOutput_ds,
	                                                     const PRISMATIC_FLOAT_PRECISION* qya_d,
	                                                     const PRISMATIC_FLOAT_PRECISION* qxa_d,
	                                                     const PRISMATIC_CUDA_COMPLEX_FLOAT* prop_d,
	                                                     const size_t Nstart,
	                                                     const size_t Nstop,
	                                                     const size_t dimj,
	                                                     const size_t dimi,
	                                                     const PRISMATIC_FLOAT_PRECISION* alphaInd_d,
	                                                     const hipfftHandle& plan,
	                                                     hipStream_t& stream){

		// initialize psi
		const size_t psi_size = dimj*dimi;
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			PRISMATIC_FLOAT_PRECISION yp = pars.yp[ay];
			PRISMATIC_FLOAT_PRECISION xp = pars.xp[ax];
			initializePsi << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
			                                                (psi_ds + (batch_idx * psi_size), PsiProbeInit_d, qya_d, qxa_d, psi_size, yp, xp);
		}

		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {

			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*psi_size], psi_size * sizeof(PRISMATIC_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
				                                                   (psi_ds + (batch_idx * psi_size), trans_d, psi_size);
			}
			cufftErrchk(PRISMATIC_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
				                                                   (psi_ds + (batch_idx * psi_size), prop_d, psi_size);
				divide_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
				                                                 (psi_ds + (batch_idx * psi_size), PRISMATIC_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
			}
		}
		abs_squared << < (psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			formatOutput_GPU_integrate(pars, psiIntensity_ds + (batch_idx * psi_size),
			                           alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		}
		if (pars.meta.save4DComplexOutput) {
            real << < (psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
            for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
                const size_t ay = (Nstart + batch_idx) / pars.xp.size();
                const size_t ax = (Nstart + batch_idx) % pars.xp.size();
                formatOutput_GPU_real(pars, psiIntensity_ds + (batch_idx * psi_size),
                                        alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
            }
            imag << < (psi_size*(Nstop-Nstart) - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psiIntensity_ds, psi_ds, psi_size*(Nstop-Nstart));
            for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
                const size_t ay = (Nstart + batch_idx) / pars.xp.size();
                const size_t ax = (Nstart + batch_idx) % pars.xp.size();
                formatOutput_GPU_imag(pars, psiIntensity_ds + (batch_idx * psi_size),
                                        alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
            }
        }
	}
    __host__ void buildMultisliceOutput_GPU_singlexfer(Parameters <PRISMATIC_FLOAT_PRECISION> &pars){
		using namespace std;
#ifdef PRISMATIC_BUILDING_GUI
	    pars.progressbar->signalDescriptionMessage("Computing final output (Multislice)");
#endif
		CudaParameters<PRISMATIC_FLOAT_PRECISION> cuda_pars;

		// determine the batch size to use
	    pars.meta.batchSizeGPU = min(pars.meta.batchSizeTargetGPU, max((size_t)1, pars.xp.size() * pars.yp.size() / max((size_t)1, (pars.meta.numStreamsPerGPU*pars.meta.numGPUs))));

		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		createPlansAndStreamsM(pars, cuda_pars);

	    // create page-locked (pinned) host memory buffers
	    allocatePinnedHostMemory_M(pars, cuda_pars);

	    copyToPinnedMemory_M(pars, cuda_pars);

	    // allocate memory on the GPUs
	    allocateDeviceMemory_singlexferM(pars, cuda_pars);

	    // copy memory to GPUs
	    copyToGPUMemory_singlexferM(pars, cuda_pars);

	    // launch GPU and CPU workers
	    launchWorkers_singlexferM(pars, cuda_pars);

	    // free memory
	    cleanupMemoryM(pars, cuda_pars);
	}

	__host__ void buildMultisliceOutput_GPU_streaming(Parameters <PRISMATIC_FLOAT_PRECISION> &pars){
#ifdef PRISMATIC_BUILDING_GUI
		pars.progressbar->signalDescriptionMessage("Computing final output (Multislice)");
#endif
		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		using namespace std;

		CudaParameters<PRISMATIC_FLOAT_PRECISION> cuda_pars;

		// determine the batch size to use
		pars.meta.batchSizeGPU = min(pars.meta.batchSizeTargetGPU, max((size_t)1, pars.xp.size() * pars.yp.size() / max((size_t)1, (pars.meta.numStreamsPerGPU*pars.meta.numGPUs))));

		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		createPlansAndStreamsM(pars, cuda_pars);

		allocatePinnedHostMemory_M(pars, cuda_pars);

		copyToPinnedMemory_M(pars,cuda_pars);

		// allocate memory on the GPUs
		allocateDeviceMemory_streamingM(pars, cuda_pars);

		// copy memory to GPUs
		copyToGPUMemory_streamingM(pars, cuda_pars);

		// launch GPU and CPU workers
		launchWorkers_streamingM(pars, cuda_pars);

		// free memory
		cleanupMemoryM(pars, cuda_pars);
	}
}
