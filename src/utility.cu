#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// Prismatic is distributed under the GNU General Public License (GPL)
// If you use Prismatic, we kindly ask that you cite the following papers:

// 1. Ophus, C.: A fast image simulation algorithm for scanning
//    transmission electron microscopy. Advanced Structural and
//    Chemical Imaging 3(1), 13 (2017)

// 2. Pryor, Jr., A., Ophus, C., and Miao, J.: A Streaming Multi-GPU
//    Implementation of Image Simulation Algorithms for Scanning
//	  Transmission Electron Microscopy. arXiv:1706.08563 (2017)

#include "utility.cuh"
#include "hip/hip_complex.h"
#include <iostream>

#define PI 3.14159265359
// define some constants
__device__ __constant__ float pi_f                  = PI;
__device__ __constant__ hipFloatComplex i_f          = {0, 1};
__device__ __constant__ hipFloatComplex pi_cx_f      = {PI, 0};
__device__ __constant__ hipFloatComplex minus_2pii_f = {0, -2*PI};
__device__ __constant__ double pi                   = PI;
__device__ __constant__ hipDoubleComplex i           = {0, 1};
__device__ __constant__ hipDoubleComplex pi_cx       = {PI, 0};
__device__ __constant__ hipDoubleComplex minus_2pii  = {0, -2*PI};

//atomicAdd for doubles on devices with compute capability < 6. This is directly copied from the CUDA Programming Guide
#if __CUDA_ARCH__ < 600
__device__  double atomicAdd_double(double* address, const double val)
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
		                __double_as_longlong(val +
		                                     __longlong_as_double(assumed)));

//		 Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif



// computes exp(real(a) + i * imag(a))
__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
	double e = exp(a.x);
	double s,c;
	sincos(a.y, &s, &c);
	return make_hipDoubleComplex(e*c, e*s);
}
__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
	float e = expf(a.x);
	float s,c;
	sincosf(a.y, &s, &c);
	return make_hipFloatComplex(e*c, e*s);
}

__global__ void initializePsi_oneNonzero(hipFloatComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipFloatComplex(1,0):make_hipFloatComplex(0,0);
	}
}

__global__ void initializePsi_oneNonzero(hipDoubleComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipDoubleComplex(1,0):make_hipDoubleComplex(0,0);
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipDoubleComplex* arr,
                                 const hipDoubleComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex a = arr[idx];
		hipDoubleComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipFloatComplex* arr,
                                 const hipFloatComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex a = arr[idx];
		hipFloatComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipDoubleComplex* arr,
                             const hipDoubleComplex* other,
                             const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
//		hipDoubleComplex a = arr[idx];
//		hipDoubleComplex o = other[idx];
		arr[idx] = hipCmul(arr[idx], other[idx]);
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipFloatComplex* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], other[idx]);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipDoubleComplex* arr,
                                      const hipDoubleComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmul(arr[idx], val);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipFloatComplex* arr,
                                      const hipFloatComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], val);
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(double* arr,
                                    const double val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(float* arr,
                                    const float val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}


//// divide two complex arrays
//__global__ void divide_inplace(PRISMATIC_CUDA_COMPLEX_FLOAT* arr,
//                               const PRISMATIC_FLOAT_PRECISION val,
//                               const size_t N){
//	int idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if (idx < N) {
//		arr[idx].x /= val;
//		arr[idx].y /= val;
//	}
//}

__global__ void divide_inplace(hipDoubleComplex* arr,
                               const hipDoubleComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdiv(arr[idx], val);
	}
}

__global__ void divide_inplace(hipFloatComplex* arr,
                               const hipFloatComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdivf(arr[idx], val);
	}
}

// set all array values to val
__global__ void setAll(double *data, double val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(float *data, float val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipDoubleComplex *psi_d,
                              const hipDoubleComplex* PsiProbeInit_d,
                              const double* qya_d,
                              const double* qxa_d,
                              const size_t N,
                              const double yp,
                              const double xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex arg;
		arg = make_hipDoubleComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmul(PsiProbeInit_d[idx], exp_cx(hipCmul(minus_2pii,arg)));
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipFloatComplex *psi_d,
                              const hipFloatComplex* PsiProbeInit_d,
                              const float* qya_d,
                              const float* qxa_d,
                              const size_t N,
                              const float yp,
                              const float xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex arg;
		arg = make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii_f,arg)));
	}
}


// compute modulus squared of other and store in arr
__global__ void abs_squared(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void abs_squared(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void real(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = re + 0.0*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void real(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = re + 0.0*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void imag(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = 0.0*re + im;
	}
}

// compute modulus squared of other and store in arr
__global__ void imag(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = 0.0*re + im;
	}
}

__global__ void array_subset(const hipDoubleComplex* psi_d,
                             hipDoubleComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < dimj_small*dimi_small) {
		int y = idx / (int)dimi_small;
		int x = idx % (int)dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}
__global__ void array_subset(const hipFloatComplex* psi_d,
                             hipFloatComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < dimj_small * dimi_small) {
		int y = idx / (int) dimi_small;
		int x = idx % (int) dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}




__global__ void shiftIndices(long* vec_out, const long by, const long imageSize, const long N){

		//int idx = threadIdx.x + blockDim.x * blockIdx.x;
		long idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			vec_out[idx] = (imageSize + ((idx - N/2 + by) % imageSize)) % imageSize;
//			vec_out[idx] =  (idx - N/2 + by) % imageSize;
//			vec_out[idx] = 0;
		}
	}

__global__ void zeroIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = vec_out[idx] - vec_out[0];
	}
}

__global__ void resetIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = idx;
	}
}


__global__ void computePhaseCoeffs(hipFloatComplex* phaseCoeffs,
                                   const hipFloatComplex *PsiProbeInit_d,
                                   const float * qyaReduce_d,
                                   const float * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const float yp,
                                   const float xp,
                                   const float yTiltShift,
                                   const float xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipFloatComplex xp_cx = make_hipFloatComplex(xp, 0);
		hipFloatComplex yp_cx = make_hipFloatComplex(yp, 0);
		hipFloatComplex xTiltShift_cx = make_hipFloatComplex(xTiltShift, 0);
		hipFloatComplex yTiltShift_cx = make_hipFloatComplex(yTiltShift, 0);
		hipFloatComplex qya = make_hipFloatComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex qxa = make_hipFloatComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex arg1 = hipCmulf(qxa, hipCaddf(xp_cx, xTiltShift_cx));
		hipFloatComplex arg2 = hipCmulf(qya, hipCaddf(yp_cx, yTiltShift_cx));
		hipFloatComplex arg = hipCaddf(arg1, arg2);
		hipFloatComplex phase_shift = exp_cx(hipCmulf(minus_2pii_f, arg));
		phaseCoeffs[idx] = hipCmulf(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}

__global__ void computePhaseCoeffs(hipDoubleComplex* phaseCoeffs,
                                   const hipDoubleComplex *PsiProbeInit_d,
                                   const double * qyaReduce_d,
                                   const double * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const double yp,
                                   const double xp,
                                   const double yTiltShift,
                                   const double xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipDoubleComplex xp_cx = make_hipDoubleComplex(xp, 0);
		hipDoubleComplex yp_cx = make_hipDoubleComplex(yp, 0);
		hipDoubleComplex xTiltShift_cx = make_hipDoubleComplex(xTiltShift, 0);
		hipDoubleComplex yTiltShift_cx = make_hipDoubleComplex(yTiltShift, 0);
		hipDoubleComplex qya = make_hipDoubleComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex qxa = make_hipDoubleComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex arg1 = hipCmul(qxa, hipCadd(xp_cx, xTiltShift_cx));
		hipDoubleComplex arg2 = hipCmul(qya, hipCadd(yp_cx, yTiltShift_cx));
		hipDoubleComplex arg = hipCadd(arg1, arg2);
		hipDoubleComplex phase_shift = exp_cx(hipCmul(minus_2pii, arg));
		phaseCoeffs[idx] = hipCmul(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}





// integrate computed intensities radially
__global__ void integrateDetector(const float* psiIntensity_ds,
                                  const float* alphaInd_d,
                                  float* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			//atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
			atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}

__global__ void integrateDetector(const double* psiIntensity_ds,
                                  const double* alphaInd_d,
                                  double* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			//atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
			atomicAdd_double(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}


void formatOutput_GPU_integrate(Prismatic::Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
                                PRISMATIC_FLOAT_PRECISION *psiIntensity_ds,
                                const PRISMATIC_FLOAT_PRECISION *alphaInd_d,
                                PRISMATIC_FLOAT_PRECISION *output_ph,
                                PRISMATIC_FLOAT_PRECISION *integratedOutput_ds,
                                const size_t ay,
                                const size_t ax,
                                const size_t& dimj,
                                const size_t& dimi,
                                const hipStream_t& stream,
                                const long& scale) {

	//save 4D output if applicable
	if (pars.meta.save4DOutput) {
		// This section could be improved. It currently makes a new 2D array, copies to it, and
		// then saves the image. This allocates arrays multiple times unneccessarily, and the allocated
		// memory isn't pinned, so the memcpy is not asynchronous.
		std::string section4DFilename = generateFilename(pars, ay, ax);
		Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> currentImage = Prismatic::zeros_ND<2, PRISMATIC_FLOAT_PRECISION>(
				{{pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi()}});
		cudaErrchk(hipMemcpyAsync(&currentImage[0],
		                           psiIntensity_ds,
		                           pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION),
		                           hipMemcpyDeviceToHost,
		                           stream));
		currentImage.toMRC_f(section4DFilename.c_str());
	}
	
	
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);


	size_t num_integration_bins = pars.detectorAngles.size();
	setAll << < (num_integration_bins - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
	                                                                            (integratedOutput_ds, 0, num_integration_bins);
//	if (ax == 0 & ay == 0) {
//		PRISMATIC_FLOAT_PRECISION ans;
//		for (auto i = 0; i < pars.detectorAngles.size(); ++i) {
//			hipMemcpy(&ans, integratedOutput_ds + i, sizeof(ans), hipMemcpyDeviceToHost);
//			std::cout << "set 0 integratedOutput_ds[" << i << "] = " << ans << std::endl;
//
//		}
//	}
//
//	if (ax == 0 & ay == 0) {
//		PRISMATIC_FLOAT_PRECISION ans;
//		for (auto i = 0; i < pars.detectorAngles.size(); ++i) {
//			hipMemcpy(&ans, alphaInd_d + i, sizeof(ans), hipMemcpyDeviceToHost);
//			std::cout << "alphaInd_d[" << i << "] = " << ans << std::endl;
//
//		}
//	}
//
//	if (ax == 0 & ay == 0) {
//		PRISMATIC_FLOAT_PRECISION ans;
//		for (auto i = 98; i < pars.detectorAngles.size(); ++i) {
//			hipMemcpy(&ans, psiIntensity_ds + i, sizeof(ans), hipMemcpyDeviceToHost);
//			std::cout << "psiIntensity_ds[" << i << "] = " << ans << std::endl;
//
//		}
//	}
	integrateDetector << < (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
	                                                                              (psiIntensity_ds, alphaInd_d, integratedOutput_ds,
			                                                                              dimj *
			                                                                              dimi, num_integration_bins);
//	if (ax == 0 & ay == 0) {
//		PRISMATIC_FLOAT_PRECISION ans;
//		for (auto i = 97; i < pars.detectorAngles.size(); ++i) {
//			hipMemcpy(&ans, integratedOutput_ds + i, sizeof(ans), hipMemcpyDeviceToHost);
//			std::cout << "after integrate integratedOutput_ds[" << i << "] = " << ans << std::endl;
//
//		}
//	}
//	if (scale != 1) {
//		if (ax==0 & ay==0)std::cout << "scale = " << scale << std::endl;
	multiply_arr_scalar << < (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
	                                                                                (integratedOutput_ds, scale, num_integration_bins);
//	}
////	integrateDetector<<< (dimj*dimi - 1)/BLOCK_SIZE1D + 1, BLOCK_SIZE1D, sizeof(PRISMATIC_FLOAT_PRECISION) * pars.detectorAngles.size(), stream>>>(psiIntensity_ds, alphaInd_d, dimj*dimi, num_integration_bins);

	// Copy result. For the integration case the 4th dim of stack is 1, so the offset strides need only consider k and j
	cudaErrchk(hipMemcpyAsync(output_ph, integratedOutput_ds,
	                           num_integration_bins * sizeof(PRISMATIC_FLOAT_PRECISION),
	                           hipMemcpyDeviceToHost, stream));

//	 wait for the copy to complete and then copy on the host. Other host threads exist doing work so this wait isn't costing anything
	cudaErrchk(hipStreamSynchronize(stream));
	//const size_t stack_start_offset = ay*pars.output.get_dimk()*pars.output.get_dimj()+ ax*pars.output.get_dimj();
	const size_t stack_start_offset =
			ay * pars.output.get_dimj() * pars.output.get_dimi() + ax * pars.output.get_dimi();
	memcpy(&pars.output[stack_start_offset], output_ph, num_integration_bins * sizeof(PRISMATIC_FLOAT_PRECISION));
}

void formatOutput_GPU_real(Prismatic::Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
                                PRISMATIC_FLOAT_PRECISION *psiIntensity_ds,
                                const PRISMATIC_FLOAT_PRECISION *alphaInd_d,
                                PRISMATIC_FLOAT_PRECISION *output_ph,
                                PRISMATIC_FLOAT_PRECISION *integratedOutput_ds,
                                const size_t ay,
                                const size_t ax,
                                const size_t& dimj,
                                const size_t& dimi,
                                const hipStream_t& stream,
                                const long& scale) {

		// This section could be improved. It currently makes a new 2D array, copies to it, and
		// then saves the image. This allocates arrays multiple times unneccessarily, and the allocated
		// memory isn't pinned, so the memcpy is not asynchronous.
		std::string section4DFilename = "Complex_real_";
        section4DFilename += generateFilename(pars, ay, ax);
		Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> currentImage = Prismatic::zeros_ND<2, PRISMATIC_FLOAT_PRECISION>(
				{{pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi()}});
		cudaErrchk(hipMemcpyAsync(&currentImage[0],
		                           psiIntensity_ds,
		                           pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION),
		                           hipMemcpyDeviceToHost,
		                           stream));
		currentImage.toMRC_f(section4DFilename.c_str());
}

void formatOutput_GPU_imag(Prismatic::Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
                                PRISMATIC_FLOAT_PRECISION *psiIntensity_ds,
                                const PRISMATIC_FLOAT_PRECISION *alphaInd_d,
                                PRISMATIC_FLOAT_PRECISION *output_ph,
                                PRISMATIC_FLOAT_PRECISION *integratedOutput_ds,
                                const size_t ay,
                                const size_t ax,
                                const size_t& dimj,
                                const size_t& dimi,
                                const hipStream_t& stream,
                                const long& scale) {

		// This section could be improved. It currently makes a new 2D array, copies to it, and
		// then saves the image. This allocates arrays multiple times unneccessarily, and the allocated
		// memory isn't pinned, so the memcpy is not asynchronous.
		std::string section4DFilename = "Complex_imag_";
        section4DFilename += generateFilename(pars, ay, ax);
		Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> currentImage = Prismatic::zeros_ND<2, PRISMATIC_FLOAT_PRECISION>(
				{{pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi()}});
		cudaErrchk(hipMemcpyAsync(&currentImage[0],
		                           psiIntensity_ds,
		                           pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION),
		                           hipMemcpyDeviceToHost,
		                           stream));
		currentImage.toMRC_f(section4DFilename.c_str());
}
















size_t getNextPower2(const size_t& val){
	size_t p = 0;
	while (pow(2,p) <= val)++p;
	return p;
}
